#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>

#include "stdio.h"
#include <cmath>
#include <hip/hip_runtime.h>

#include "kernel.h"


__device__
float angle(float x, float y) 
{ 
	float theta = std::atan2(x, y); 
	theta *= 180.0f / 3.14f; 
	return theta - 90;
}


__global__
void MyKernel1(float2* d_p, float2* d_v, float* d_angle,float2 mouseLocation, 
    unsigned int* indexes, unsigned int* begins, unsigned int* ends,
    int howMany, int GridDim, float WIDTH, bool MouseMode, float allignCoef, float groupCoef, float separateCoef, float timeElapsed ) {

    //give boid appropiate perception distances
    float allignDistance = 0.08f;
    float groupDistance = 0.08f;
    float separateDistance = 0.03f;
    if (howMany <= 2000)
    {
        allignDistance = 0.24f;
        groupDistance = 0.24f;
        separateDistance = 0.1f;
    }

    //calculate boid index 
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= howMany) return;
    const int ii = indexes[i];

    //calculate boid bucket
    unsigned int x = ((d_p[ii].x + WIDTH) / (WIDTH * 2)) *GridDim;
    unsigned int y = ((d_p[ii].y + HEIGHT) / (HEIGHT * 2)) * GridDim;
    int bucket =  y * GridDim + x;

    
    //discard neighbour buckets that boid cannot see       //  __________
    bool b2 = (bucket / GridDim != GridDim - 1);           //  |b7 b8 b9|   
    bool b4 = (bucket % GridDim != 0);                     //  |b4    b6|
    bool b6 = (bucket % GridDim != GridDim - 1);           //  |b1 b2 b3|
    bool b8 = (bucket / GridDim != 0);                     //  ~~~~~~~~~~

    b6 = (b6 && ((x + 1) * (2 * WIDTH / (GridDim)) - (d_p[ii].x + WIDTH) <= allignDistance));
    b4 = (b4 && ((d_p[ii].x + WIDTH) - (x * (2 * WIDTH / GridDim)) <= allignDistance));
    b8 = (b8 && (((y + 1) * 2 * HEIGHT / (GridDim)) - (d_p[ii].y + HEIGHT) <= allignDistance));
    b2 = (b2 && ((d_p[ii].y + HEIGHT) - (y * (2 * HEIGHT / GridDim)) <= allignDistance));

    bool b1 = (b2 && b4);
    bool b3 = (b6 && b2);
    bool b7 = (b8 && b4);
    bool b9 = (b8 && b6);

#pragma region collect_data
    //collect neighbours data
    float vxAllign = 0, vyAllign = 0, dxGroup = 0, dyGroup = 0, dxSeparate = 0, dySeparate = 0, distance;
    int counterAllign = 0, counterGroup = 0, counterSeparate = 0;

        for (int j = begins[bucket]; j < ends[bucket]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance) 
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance) 
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f) 
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }       
        }

        if (b1) for (int j = begins[bucket - GridDim - 1]; j < ends[bucket - GridDim - 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b2) for (int j = begins[bucket - GridDim]; j < ends[bucket - GridDim]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b3) for (int j = begins[bucket - GridDim + 1]; j < ends[bucket - GridDim + 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b4) for (int j = begins[bucket - 1]; j < ends[bucket - 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b6) for (int j = begins[bucket + 1]; j < ends[bucket + 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b7) for (int j = begins[bucket + GridDim - 1]; j < ends[bucket + GridDim - 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b8) for (int j = begins[bucket + GridDim]; j < ends[bucket + GridDim]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

        if (b9) for (int j = begins[bucket + GridDim + 1]; j < ends[bucket + GridDim + 1]; ++j)
        {
            int jj = indexes[j];
            if (ii == jj) continue;
            distance = sqrtf(((d_p[jj].x - d_p[ii].x) * (d_p[jj].x - d_p[ii].x)) + ((d_p[jj].y - d_p[ii].y) * (d_p[jj].y - d_p[ii].y)));
            if (distance < allignDistance)
            {
                vxAllign += (d_v[jj].x);
                vyAllign += (d_v[jj].y);
                counterAllign++;
            }
            if (distance < groupDistance)
            {
                dxGroup += d_p[jj].x;
                dyGroup += d_p[jj].y;
                counterGroup++;
            }
            if (distance < separateDistance && distance > 0.0001f)
            {
                dxSeparate -= ((d_p[jj].x - d_p[ii].x) / (distance));
                dySeparate -= ((d_p[jj].y - d_p[ii].y) / (distance));
                counterSeparate++;
            }
        }

#pragma endregion collect_data

        float ax = 0;
        float ay = 0;

        // check if boid is near mouse cursor
        float dist = sqrtf(((mouseLocation.x - d_p[ii].x) * (mouseLocation.x - d_p[ii].x)) + ((mouseLocation.y - d_p[ii].y) * (mouseLocation.y - d_p[ii].y)));
        if (MouseMode && dist < MOUSE_RADIUS)
        {
            ax = ((d_p[ii].x - mouseLocation.x) / dist) * MOUSE_FORCE;
            ay = ((d_p[ii].y - mouseLocation.y) / dist) * MOUSE_FORCE;

            if (dist > 0.9f * MOUSE_RADIUS)
            {
                ax = -d_v[ii].x * 0.5f;
                ay = -d_v[ii].x * 0.5f;
            }
        }
        //Calculate acceleration
        else
        {
           if (counterAllign)
           {
               vxAllign /= counterAllign;
               vyAllign /= counterAllign;
               vxAllign *= allignCoef;
               vyAllign *= allignCoef;
               ax += (vxAllign - d_v[ii].x);
               ay += (vyAllign - d_v[ii].y);
           }
           if (counterGroup)
           {
               dxGroup /= (counterGroup);
               dyGroup /= (counterGroup);
               dxGroup -= d_p[ii].x;
               dyGroup -= d_p[ii].y;
               dxGroup *= groupCoef;
               dyGroup *= groupCoef;
               ax += (dxGroup);
               ay += (dyGroup);
           }
           if (counterSeparate)
           {
               dxSeparate /= counterSeparate;
               dySeparate /= counterSeparate;
               dxSeparate *= separateCoef;
               dySeparate *= separateCoef;
               ax += (dxSeparate);
               ay += (dySeparate);
           }   

        }

        //speed up slow boids
        if (fabs(d_v[ii].x) < 0.1f && fabs(d_v[ii].y) < 0.1f) 
        {
            d_v[ii].x *= 1.03f;
            d_v[ii].y *= 1.03f;
        }
        //water resistance slows fishes
        else 
        {
            d_v[ii].x *= 0.99f;
            d_v[ii].y *= 0.99f;
        }

        //speed up boids and change their positions
        d_v[ii].x += ax * timeElapsed;
        d_v[ii].y += ay * timeElapsed;

        d_p[ii].x += d_v[ii].x * timeElapsed;
        d_p[ii].y += d_v[ii].y * timeElapsed;

        //calculate boid swimming angle
        d_angle[ii] = angle(d_v[ii].y, d_v[ii].x);

        //keep boids in displaying area
        if (d_p[ii].x >= WIDTH) d_p[ii].x = -WIDTH + 0.002f;
        else if (d_p[ii].x <= -WIDTH) d_p[ii].x = WIDTH - 0.002f;
        if (d_p[ii].y >= HEIGHT) d_p[ii].y = -HEIGHT + 0.002f;
        else if (d_p[ii].y <= -HEIGHT) d_p[ii].y = HEIGHT - 0.002f;
}




struct calculate_bucket_index
{
    float w, h; 
    int gridDim;
    __host__ __device__
        calculate_bucket_index(float width, float height, int dim)
        :w(width), h(height), gridDim(dim) {}
    __host__ __device__
     unsigned int operator()(float2 p) const
    {
        // coordinates of the grid cell containing point p
        unsigned int x = ((p.x + w) / (w * 2)) * gridDim;                               
        unsigned int y = ((p.y + h) / (h * 2)) * gridDim;
        // return the bucket's linear index
        return y * gridDim + x;
    }
};


//Calculate new speeds and positions of each boid
void boidsLauncher(Flock* flock, float2 pos, int howMany,float WIDTH, bool MouseMode,float alignCoef,float groupCoef,float separateCoef,float timeElapsed, unsigned int gridDim)
{
    hipError_t err = hipSuccess;

    //Copy memory to device
    thrust::device_vector<float2> d_p = flock->p;
    thrust::device_vector<float2> d_v = flock->v;
    thrust::device_vector<float> d_angle = flock->angle;
    thrust::device_vector<unsigned int> sorted_indexes = flock->index;
 
    //bucket sort using thrust library
    //give each boid its bucket index 
    thrust::device_vector<unsigned int> bucket_indices(howMany);
    thrust::transform(d_p.begin(), d_p.end(), bucket_indices.begin(), calculate_bucket_index(WIDTH, HEIGHT, gridDim));

    //sort boid indexes by bucket
    thrust::sort_by_key(bucket_indices.begin(), bucket_indices.end(), sorted_indexes.begin());

    //return each bucket begin and end
    thrust::device_vector<unsigned int> bucket_start(gridDim * gridDim);
    thrust::device_vector<unsigned int> bucket_end(gridDim * gridDim);

    thrust::counting_iterator<unsigned int> search_begin(0);
    thrust::lower_bound(bucket_indices.begin(), bucket_indices.end(), search_begin, search_begin + gridDim * gridDim, bucket_start.begin());  //returns iterator on first boid with bucket index => i
    thrust::upper_bound(bucket_indices.begin(), bucket_indices.end(), search_begin, search_begin + gridDim * gridDim, bucket_end.begin());  //returns iterator on first boid with bucket index > i 


    //cast thrust vectors to raw_pointers to use them in kernel
    float2* ptr_p = thrust::raw_pointer_cast(&d_p[0]);
    float2* ptr_v = thrust::raw_pointer_cast(&d_v[0]);
    float* ptr_angle = thrust::raw_pointer_cast(&d_angle[0]);
    unsigned int* ptr_starts = thrust::raw_pointer_cast(&bucket_start[0]);
    unsigned int* ptr_ends = thrust::raw_pointer_cast(&bucket_end[0]);
    unsigned int* ptr_sorted_indexes = thrust::raw_pointer_cast(&sorted_indexes[0]);


    //run kernel 
    int threadsPerBlock = 256;
    int blocksPerGrid = (howMany + threadsPerBlock - 1) / threadsPerBlock;
    MyKernel1 <<<blocksPerGrid,  threadsPerBlock>> > (ptr_p, ptr_v, ptr_angle, pos, ptr_sorted_indexes, ptr_starts, ptr_ends, 
                                                        howMany, gridDim, WIDTH, MouseMode, alignCoef, groupCoef, separateCoef, timeElapsed);  

    //check errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch boids kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //copy memory from device to host
    flock->p = d_p;
    flock->v = d_v;
    flock->angle = d_angle;
}

